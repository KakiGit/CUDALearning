#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>


// Kernel function to increment each element in the array
__global__ void increment(int* a, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        a[idx] += 1;
    }
}

hipError_t runGraphs() {
    printf("starting %s\n", __func__);

    hipError_t cudaStatus;
    int n = 1000;
    int size = n * sizeof(int);
    int* d_a;
    hipGraph_t graph;
    hipGraphExec_t instance;
    hipStream_t stream;

    // Allocate memory on the device (GPU)
    hipMalloc((void**)&d_a, size);
    hipMemset(d_a, 0, size);

    // Define the number of threads per block and the number of blocks
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Create a CUDA stream
    hipStreamCreate(&stream);

    // Capture the graph
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    increment<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_a, n);
    hipStreamEndCapture(stream, &graph);

    // Instantiate the graph
    hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

    // Launch the graph multiple times
    int launchCount = 5;
    for (int i = 0; i < launchCount; ++i) {
        hipGraphLaunch(instance, stream);
    }

    // Wait for the graph executions to complete
    hipStreamSynchronize(stream);

    // Copy result back to the host to verify
    int* h_a = (int*)malloc(size);
    hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);

    // Print the result
    printf("Result after launching the graph %d times:\n", launchCount);
    for (int i = 0; i < 10; i++) { // Print the first 10 elements
        printf("%d ", h_a[i]);
    }
    printf("\n");

    // Clean up
    hipGraphDestroy(graph);
    hipGraphExecDestroy(instance);
    hipStreamDestroy(stream);
    hipFree(d_a);
    free(h_a);
    cudaStatus = hipGetLastError();
    return cudaStatus;
}