#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>

#define MASK_WIDTH 3
#define TILE_WIDTH 16

__global__ void convolutionKernel(float* input, float* output, float* mask, int width, int height) {
	__shared__ float tile[TILE_WIDTH + MASK_WIDTH - 1][TILE_WIDTH + MASK_WIDTH - 1];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row_o = blockIdx.y * TILE_WIDTH + ty;
	int col_o = blockIdx.x * TILE_WIDTH + tx;
	// Input index is 1 pixel shift than output index
	// So output[0][0] -> tile[1][1] neighbors -> input[0][0]
	// There are some edge cases but this is good enough.
	int row_i = row_o - 1;
	int col_i = col_o - 1;

	if ((row_i >= 0) && (row_i < height) && (col_i >= 0) && (col_i < width)) {
		tile[ty][tx] = input[row_i * width + col_i];
	}
	else {
		tile[ty][tx] = 0.0f;
	}

	__syncthreads();

	float output_value = 0.0f;
	for (int i = 0; i < MASK_WIDTH; ++i) {
		for (int j = 0; j < MASK_WIDTH; ++j) {
			output_value += tile[ty + i][tx + j] * mask[i * MASK_WIDTH + j];
		}
	}

	if (row_o < height && col_o < width) {
		output[row_o * width + col_o] = output_value > 0.0f ? output_value : 0.0f;
	}
}

void convolution(float* input, float* output, float* mask, int width, int height) {
	int size = width * height * sizeof(float);
	float* d_input, * d_output, * d_mask;

	hipMalloc((void**)&d_input, size);
	hipMalloc((void**)&d_output, size);
	hipMalloc((void**)&d_mask, MASK_WIDTH * MASK_WIDTH * sizeof(float));

	hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);
	hipMemcpy(d_mask, mask, MASK_WIDTH * MASK_WIDTH * sizeof(float), hipMemcpyHostToDevice);

	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
	dim3 dimGrid((width + TILE_WIDTH - 1) / TILE_WIDTH, (height + TILE_WIDTH - 1) / TILE_WIDTH);
	convolutionKernel<<<dimGrid, dimBlock>>>(d_input, d_output, d_mask, width, height);

	hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

	hipFree(d_input);
	hipFree(d_output);
	hipFree(d_mask);
}

hipError_t cudaImageConvolution() {
	printf("starting %s\n", __func__);

	hipError_t cudaStatus;
	int width = 1024;
	int height = 1024;
	int size = width * height * sizeof(float);

	float* input = (float*)malloc(size);
	float* output = (float*)malloc(size);
	float unsharp_kernel[MASK_WIDTH * MASK_WIDTH] = {
		0, -1, 0,
		-1, 5, -1,
		0, -1, 0
	};

	// Initialize input image
	for (int i = 0; i < width * height; ++i) {
		input[i] = (float)(i % 256);
	}

	convolution(input, output, unsharp_kernel, width, height);

	// Output some values for verification
	for (int i = 0; i < 10; ++i) {
		printf("%f ", output[i]);
	}
	printf("\n");

	free(input);
	free(output);

	cudaStatus = hipGetLastError();
	return cudaStatus;
}