﻿
#include "hip/hip_runtime.h"
#include ""
#include "functions.h"
#include <stdio.h>


int main()
{

    hipError_t cudaStatus;

    //addWithCuda();
    //matrixMulWithCuda();
    //runGraphs();
    //runCooperativeGroups();
    //cudaImageConvolution();
    cudaTrainNeuralNetwork();
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}